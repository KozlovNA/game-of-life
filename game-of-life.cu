#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <array>
#include <fstream>
#include <string>

using namespace std;

#define CUDA_FLOAT float
#define BLOCKX_SIZE 4
#define BLOCKY_SIZE 4
#define BLOCKZ_SIZE 4
#define GRIDX_SIZE 8
#define GRIDY_SIZE 8
#define GRIDZ_SIZE 8

__global__ void life_kern(int *field, int *tmp_field)
{   
    int tx = threadIdx.x + blockIdx.x * BLOCKX_SIZE;
    int ty = threadIdx.y + blockIdx.y * BLOCKY_SIZE;    //indexing threads
    int tz = threadIdx.z + blockIdx.z * BLOCKZ_SIZE; 
    int S = 0;
    //printf("%d, %d, %d, %d\n", tmp_field[tx + (ty)*GRIDX_SIZE*BLOCKX_SIZE + (tz)*GRIDX_SIZE*BLOCKX_SIZE*GRIDY_SIZE*BLOCKY_SIZE], tx, ty, tz);

    for (int i = -1; i < 2; i+=1)
    {
        for (int j = -1; j < 2; j+=1)
        {
            for (int k = -1; k < 2; k+=1)
            {   
                S += tmp_field[tx+i + (ty+j)*GRIDX_SIZE*BLOCKX_SIZE + (tz+k)*GRIDX_SIZE*BLOCKX_SIZE*GRIDY_SIZE*BLOCKY_SIZE];        //number of neighbours
            }   
        }    
    }
    S-=tmp_field[tx + (ty)*GRIDX_SIZE*BLOCKX_SIZE + (tz)*GRIDX_SIZE*BLOCKX_SIZE*GRIDY_SIZE*BLOCKY_SIZE];
    //if (tx==1&&ty==0&&tz==2)
    //printf("%d, %d, %d, %d\n", S, tx, ty, tz);
    if (((tmp_field[tx + (ty)*GRIDX_SIZE*BLOCKX_SIZE + (tz)*GRIDX_SIZE*BLOCKX_SIZE*GRIDY_SIZE*BLOCKY_SIZE]==1)&&(4<=S)&&(S<=7))
        ||((tmp_field[tx + (ty)*GRIDX_SIZE*BLOCKX_SIZE + (tz)*GRIDX_SIZE*BLOCKX_SIZE*GRIDY_SIZE*BLOCKY_SIZE]==0)&&(6<=S)&&(S<=7)))
    {                                                                                                                        //deciding life or death
        field[tx + (ty)*GRIDX_SIZE*BLOCKX_SIZE + (tz)*GRIDX_SIZE*BLOCKX_SIZE*GRIDY_SIZE*BLOCKY_SIZE] = 1;
    } 
    else
    {
        field[tx + (ty)*GRIDX_SIZE*BLOCKX_SIZE + (tz)*GRIDX_SIZE*BLOCKX_SIZE*GRIDY_SIZE*BLOCKY_SIZE] = 0;
    }
    
    if (tx==0||tx==GRIDX_SIZE*BLOCKX_SIZE-1||ty==0||ty==GRIDY_SIZE*BLOCKY_SIZE-1||tz==0||tz==GRIDZ_SIZE*BLOCKZ_SIZE-1)
    {
        field[tx + (ty)*GRIDX_SIZE*BLOCKX_SIZE + (tz)*GRIDX_SIZE*BLOCKX_SIZE*GRIDY_SIZE*BLOCKY_SIZE] = 0;                   //filling buffer
    }
    
    //printf("%i %i %i %i\n",field[tx + (ty)*GRIDX_SIZE*BLOCKX_SIZE + (tz)*GRIDX_SIZE*BLOCKX_SIZE*GRIDY_SIZE*BLOCKY_SIZE] = 1, tx, ty, tz);
}

void writen(int* field, int x, int y, int z)
{
    field[x + (y)*GRIDX_SIZE*BLOCKX_SIZE + (z)*GRIDX_SIZE*BLOCKX_SIZE*GRIDY_SIZE*BLOCKY_SIZE] = 1;
}

int main(int argc, char **argv)
{   
    printf("[game-of-life] - Starting\n");
    const int xs = GRIDX_SIZE*BLOCKX_SIZE;
    const int ys = GRIDY_SIZE*BLOCKY_SIZE;
    const int zs = GRIDZ_SIZE*BLOCKZ_SIZE; 
    int* field;
    int* tmp_field;                    //initiating
    int* d_field;
    int* d_tmp_field;
    int* test;
    dim3 field_size(xs, ys, zs);
    
    field = (int*)calloc ((GRIDX_SIZE*BLOCKX_SIZE)*(GRIDY_SIZE*BLOCKY_SIZE)*(GRIDZ_SIZE*BLOCKZ_SIZE), sizeof(int));          //allocating
    tmp_field = (int*)calloc ((GRIDX_SIZE*BLOCKX_SIZE)*(GRIDY_SIZE*BLOCKY_SIZE)*(GRIDZ_SIZE*BLOCKZ_SIZE), sizeof(int));
    
    
    //cudaMalloc ((void**) &test, sizeof(int));

    writen(field, 5, 7, 5);
    writen(field, 5, 8, 5);
    writen(field, 6, 8, 5);
    writen(field, 7, 5, 5);
    writen(field, 8, 5, 5);          //start field configuration
    writen(field, 8, 6, 5);
    writen(field, 5, 7, 6);
    writen(field, 5, 8, 6);
    writen(field, 6, 8, 6);
    writen(field, 7, 5, 6);
    writen(field, 8, 5, 6);
    writen(field, 8, 6, 6);
    /*
    writen(field, 8, 6, 5);
    writen(field, 8, 6, 7);
    writen(field, 8, 4, 6);
    */
    ofstream out;
    out.open("/home/starman/CUDA/game-of-life/visualisation/lnx64-compiled/data.txt");
    if (out.is_open())
    {   
        for(int i = 0; i < xs; i++)
        {
            for(int j = 0; j < ys; j++)
            {
                for(int k = 0; k < zs; k++)
                {   
                    if (field[i + (j)*GRIDX_SIZE*BLOCKX_SIZE + (k)*GRIDX_SIZE*BLOCKX_SIZE*GRIDY_SIZE*BLOCKY_SIZE]!=0)
                        out << field[i + (j)*GRIDX_SIZE*BLOCKX_SIZE + (k)*GRIDX_SIZE*BLOCKX_SIZE*GRIDY_SIZE*BLOCKY_SIZE] << ' ' << i << ' ' << j << ' ' << k << '\n';
                }
            }       
        }
        out << 0 << ' ' << 0 << ' ' << 0 << ' ' << 0 << ' '<< '\n'; 
    }

    hipMalloc ((void **) &d_tmp_field, (GRIDX_SIZE*BLOCKX_SIZE)*(GRIDY_SIZE*BLOCKY_SIZE)*(GRIDZ_SIZE*BLOCKZ_SIZE)*sizeof(int)); 
    hipMalloc ((void **) &d_field, (GRIDX_SIZE*BLOCKX_SIZE)*(GRIDY_SIZE*BLOCKY_SIZE)*(GRIDZ_SIZE*BLOCKZ_SIZE)*sizeof(int));
    hipMemcpy(d_field, field, (GRIDX_SIZE*BLOCKX_SIZE)*(GRIDY_SIZE*BLOCKY_SIZE)*(GRIDZ_SIZE*BLOCKZ_SIZE)*sizeof(int) ,hipMemcpyHostToDevice);
    hipMemcpy(d_tmp_field, field, (GRIDX_SIZE*BLOCKX_SIZE)*(GRIDY_SIZE*BLOCKY_SIZE)*(GRIDZ_SIZE*BLOCKZ_SIZE)*sizeof(int) ,hipMemcpyHostToDevice);                    //allocating device memory
    //cudaMemcpy(d_field, d_tmp_field, (GRIDX_SIZE*BLOCKX_SIZE)*(GRIDY_SIZE*BLOCKY_SIZE)*(GRIDZ_SIZE*BLOCKZ_SIZE)*sizeof(int) ,cudaMemcpyDeviceToDevice);

    dim3 block = dim3(BLOCKX_SIZE, BLOCKY_SIZE, BLOCKZ_SIZE);       //grid parameters
    dim3 grid = dim3(GRIDX_SIZE, GRIDY_SIZE, GRIDZ_SIZE);

    //life_kern<<<grid, block>>> (d_field, d_tmp_field);   //kernel

    //cudaMemcpy(d_field, field, (xs)*(ys)*(zs)*sizeof(int) ,cudaMemcpyDeviceToHost);
    for (int step = 0; step < 30; step++)
    {   
        hipMemcpy(d_tmp_field, d_field, (GRIDX_SIZE*BLOCKX_SIZE)*(GRIDY_SIZE*BLOCKY_SIZE)*(GRIDZ_SIZE*BLOCKZ_SIZE)*sizeof(int) ,hipMemcpyDeviceToDevice);
        life_kern<<<grid, block>>> (d_field, d_tmp_field);
        hipMemcpy(field, d_field, (xs)*(ys)*(zs)*sizeof(int) ,hipMemcpyDeviceToHost);
        if (out.is_open())
        {   
            for(int i = 0; i < xs; i++)
            {
                for(int j = 0; j < ys; j++)
                {
                    for(int k = 0; k < zs; k++)
                    {   if (field[i + (j)*GRIDX_SIZE*BLOCKX_SIZE + (k)*GRIDX_SIZE*BLOCKX_SIZE*GRIDY_SIZE*BLOCKY_SIZE]!=0)
                            out << field[i + (j)*GRIDX_SIZE*BLOCKX_SIZE + (k)*GRIDX_SIZE*BLOCKX_SIZE*GRIDY_SIZE*BLOCKY_SIZE] << ' ' << i << ' ' << j << ' ' << k << '\n';
                    }
                }       
            }
            out << 0 << ' ' << 0 << ' ' << 0 << ' ' << 0 << ' '<< '\n'; 
        } 
    }
    out.close();

    /*
    if (out.is_open())
    {   
        for(int i = 0; i < xs; i++)
        {
            for(int j = 0; j < ys; j++)
            {
                for(int k = 0; k < zs; k++)
                {   if (field[i + (j)*GRIDX_SIZE*BLOCKX_SIZE + (k)*GRIDX_SIZE*BLOCKX_SIZE*GRIDY_SIZE*BLOCKY_SIZE]!=0)
                        out << field[i + (j)*GRIDX_SIZE*BLOCKX_SIZE + (k)*GRIDX_SIZE*BLOCKX_SIZE*GRIDY_SIZE*BLOCKY_SIZE] << ' ' << i << ' ' << j << ' ' << k << '\n';
                }
            }       
        }
        out << 0 << ' ' << 0 << ' ' << 0 << ' ' << 0 << ' '<< '\n'; 
    }
    out.close();
    */
    //for(int i = 11130; i < 12000; i++) cout << field[i] << ' ' << i << '\n'; 
    //cout << "\n"<< xs*ys*zs << " " << (xs-1) + (ys-1)*xs + (zs-1)*xs*ys;
}   